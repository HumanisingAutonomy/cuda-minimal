/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <math.h>

void randomize(float& x, float& y, float& z);

class v3
{
public:
	float x;
	float y;
	float z;
	
	v3();
	v3(float xIn, float yIn, float zIn);
	void randomize();
	__host__ __device__ void normalize();
	__host__ __device__ void scramble();

};

v3::v3()
{
	::randomize(x, y, z);
}

v3::v3(float xIn, float yIn, float zIn) : x(xIn), y(yIn), z(zIn)
{}

void v3::randomize()
{
	::randomize(x, y, z);
}

__host__ __device__ void v3::normalize()
{
	float t = sqrt(x*x + y*y + z*z);
	x /= t;
	y /= t;
	z /= t;
}

__host__ __device__ void v3::scramble()
{
	float tx = 0.317f*(x + 1.0) + y + z * x * x + y + z;
	float ty = 0.619f*(y + 1.0) + y * y + x * y * z + y + x;
	float tz = 0.124f*(z + 1.0) + z * y + x * y * z + y + x;
	x = tx;
	y = ty;
	z = tz;
}